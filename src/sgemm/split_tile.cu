#include "hip/hip_runtime.h"
//
// Created by linn on 10/14/23.
//

#include "sgemm.cuh"

// solve shared memory bank conflict
// https://blog.csdn.net/Bruce_0712/article/details/65447608
// https://blog.csdn.net/sunmc1204953974/article/details/51078818
// x: warp 在执行时以 half-warp 为单位执行，分属于不同 warp 的线程之间不会有冲突
// 执行和调度以warp为单位，存储器访问以half-warp为单位。

template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void no_share_conflict_kernel(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    const int block_row_thread = BN / TN;
    const int block_col_thread = BM / TM;
    const int thread_num = block_row_thread * block_col_thread;
    const int THREAD_TILE = TM / 4;
    // The left and uppermost element position of thread tile in block
    int start_col = blockIdx.x * BN;
    int start_row = blockIdx.y * BM;
    int tx = threadIdx.x * TN;
    int ty = threadIdx.y * TM;

    __shared__ float As[2][BK][BM];    // transpose shared A for avoid bank conflict, for double buffering
    __shared__ float Bs[2][BK][BN];    // for double buffering

    float accum[TM][TN] = {0.};

    const int load_a_cache_time = (BK * BM) / thread_num / 4;  // Each thread load 4 float
    const int load_b_cache_time = (BK * BN) / thread_num / 4;  // Each thread load 4 float

//    float load_a_cache[4 * load_a_cache_time];
    float load_a_cache[4];
//    float load_b_cache[4 * load_b_cache_time];

    A = &A[OFFSET(start_row, 0, K)]; // Set block start position
    B = &B[OFFSET(0, start_col, N)];
    C = &C[OFFSET(start_row, start_col, N)];

    int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
    int a_tile_row = thread_id / (BK / 4);
    int a_tile_col = thread_id % (BK / 4) * 4;
    int a_tile_stride = BM / load_a_cache_time;
//    printf("A tile row, col, stride %d, %d, %d", a_tile_row, a_tile_col, a_tile_stride);

    int b_tile_row = thread_id / (BN / 4);
    int b_tile_col = thread_id % (BN / 4) * 4;
    int b_tile_stride = BK / load_b_cache_time;

    float a_reg[2][TM] = {0.};  // double buffering
    float b_reg[2][TN] = {0.};  // double buffering

    int write_idx = 0;

#pragma unroll
    for (int k = 0; k < K; k += BK) {
#pragma unroll
        for (int i = 0; i < BM; i += a_tile_stride) {
            int cache_idx = i / a_tile_stride * 4;
            FETCH_FLOAT4(load_a_cache) =
                    FETCH_FLOAT4(A[OFFSET(a_tile_row + i, a_tile_col, K)]);
            // Use load_a_cache for load 4 float at a time
            // As is saved as transpose matrix
            As[write_idx][a_tile_col][a_tile_row + i] = load_a_cache[cache_idx];
            // 这里 stride = 128，有 shared memory bank 冲突
            As[write_idx][a_tile_col + 1][a_tile_row + i] = load_a_cache[cache_idx + 1];
            As[write_idx][a_tile_col + 2][a_tile_row + i] = load_a_cache[cache_idx + 2];
            As[write_idx][a_tile_col + 3][a_tile_row + i] = load_a_cache[cache_idx + 3];
        }
#pragma unroll
        for (int i = 0; i < BK; i += b_tile_stride) {
            FETCH_FLOAT4(Bs[write_idx][b_tile_row + i][b_tile_col]) =
                    FETCH_FLOAT4(B[OFFSET(b_tile_row + i, b_tile_col, N)]);
        }
        __syncthreads();
        A += BK;    // Start position of next tile block to be processed
        B += BK * N;    // Start position of next tile block to be processed

#pragma unroll
        for (int i = 0; i < BK; ++i) {
#pragma unroll
            for (int t = 0; t < THREAD_TILE; ++t) {
                FETCH_FLOAT4(a_reg[write_idx][4 * t]) =
                        FETCH_FLOAT4(As[write_idx][i][ty / THREAD_TILE + t * BM / THREAD_TILE]);
            }
#pragma unroll
            for (int t = 0; t < THREAD_TILE; ++t) {
                FETCH_FLOAT4(b_reg[write_idx][t * 4]) =
                        FETCH_FLOAT4(Bs[write_idx][i][tx / THREAD_TILE + t * BM / THREAD_TILE]);
            }
#pragma unroll
            for (int m = 0; m < TM; ++m) {
#pragma unroll
                for (int n = 0; n < TN; ++n) {
                    accum[m][n] += a_reg[write_idx][m] * b_reg[write_idx][n];
                }
            }
        }
        write_idx ^= 1;
    }

#pragma unroll
    for (int m = 0; m < TM / 2; ++m) {
        FETCH_FLOAT4(load_a_cache) = FETCH_FLOAT4(C[OFFSET(ty / 2 + m, tx / 2, N)]);
        load_a_cache[0] = alpha * accum[m][0] + beta * load_a_cache[0];
        load_a_cache[1] = alpha * accum[m][1] + beta * load_a_cache[1];
        load_a_cache[2] = alpha * accum[m][2] + beta * load_a_cache[2];
        load_a_cache[3] = alpha * accum[m][3] + beta * load_a_cache[3];
        FETCH_FLOAT4(C[OFFSET(ty / 2 + m, tx / 2, N)]) = FETCH_FLOAT4(load_a_cache);
        FETCH_FLOAT4(load_a_cache) = FETCH_FLOAT4(C[OFFSET(ty / 2 + m, tx / 2, N) + BN / 2]);
        load_a_cache[0] = alpha * accum[m][4] + beta * load_a_cache[0];
        load_a_cache[1] = alpha * accum[m][5] + beta * load_a_cache[1];
        load_a_cache[2] = alpha * accum[m][6] + beta * load_a_cache[2];
        load_a_cache[3] = alpha * accum[m][7] + beta * load_a_cache[3];
        FETCH_FLOAT4(C[OFFSET(ty / 2 + m, tx / 2, N) + BN / 2]) = FETCH_FLOAT4(load_a_cache);
    }

#pragma unroll
    for (int m = 0; m < TM / 2; ++m) {
        FETCH_FLOAT4(load_a_cache) = FETCH_FLOAT4(C[OFFSET(ty / 2 + m + BN / 2, tx / 2, N)]);
        load_a_cache[0] = alpha * accum[m + TM / 2][0] + beta * load_a_cache[0];
        load_a_cache[1] = alpha * accum[m + TM / 2][1] + beta * load_a_cache[1];
        load_a_cache[2] = alpha * accum[m + TM / 2][2] + beta * load_a_cache[2];
        load_a_cache[3] = alpha * accum[m + TM / 2][3] + beta * load_a_cache[3];
        FETCH_FLOAT4(C[OFFSET(ty / 2 + m + BN / 2, tx / 2, N)]) = FETCH_FLOAT4(load_a_cache);
        FETCH_FLOAT4(load_a_cache) = FETCH_FLOAT4(C[OFFSET(ty / 2 + m + BN / 2, tx / 2, N) + BN / 2]);
        load_a_cache[0] = alpha * accum[m + TM / 2][4] + beta * load_a_cache[0];
        load_a_cache[1] = alpha * accum[m + TM / 2][5] + beta * load_a_cache[1];
        load_a_cache[2] = alpha * accum[m + TM / 2][6] + beta * load_a_cache[2];
        load_a_cache[3] = alpha * accum[m + TM / 2][7] + beta * load_a_cache[3];
        FETCH_FLOAT4(C[OFFSET(ty / 2 + m + BN / 2, tx / 2, N) + BN / 2]) = FETCH_FLOAT4(load_a_cache);
    }
}


void test_no_share_conflict_kernel(hipblasHandle_t handle, int M, int N, int K,
                                   float alpha, float *A, float *B, float beta, float *C) {
    const int size = 16;
    const int tile_size = 8;
    const int BM = size * tile_size;
    const int BN = size * tile_size;
    const int BK = tile_size;
    const int TM = tile_size;
    const int TN = tile_size;
    dim3 block(size, size);
    dim3 grid(CEIL_DIV(N, BN), CEIL_DIV(M, BM)); // note: change M and N here
    no_share_conflict_kernel<BM, BN, BK, TM, TN><<<grid, block>>>(M, N, K, alpha, A, B, beta, C);
}


template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void tile_2d_split_kernel(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    const int block_row_thread = BN / TN;
    const int block_col_thread = BM / TM;
    const int thread_num = block_row_thread * block_col_thread;
    int num_shared_block = CEIL_DIV(K, BK);

    __shared__ float As[2][BK][BM];    // transpose shared A for avoid bank conflict, for double buffering
    __shared__ float Bs[2][BK][BN];    // for double buffering

    float accum[TM][TN] = {0.};

    const int load_a_cache_time = (BK * BM) / thread_num / 4;  // Each thread load 4 float
    const int load_b_cache_time = (BK * BN) / thread_num / 4;  // Each thread load 4 float

    float load_a_cache[4 * load_a_cache_time];
//    float load_a_cache[4];
//    float load_b_cache[4 * load_b_cache_time];

    A = &A[OFFSET(blockIdx.y * BM, 0, K)]; // Set block start position
    B = &B[OFFSET(0, blockIdx.x * BN, N)];
    C = &C[OFFSET(blockIdx.y * BM, blockIdx.x * BN, N)];

    int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
    int a_tile_row = thread_id / (BK / 4);
    int a_tile_col = thread_id % (BK / 4) * 4;
    int a_tile_stride = BM / load_a_cache_time;
//    printf("A tile row, col, stride %d, %d, %d", a_tile_row, a_tile_col, a_tile_stride);

    int b_tile_row = thread_id / (BN / 4);
    int b_tile_col = thread_id % (BN / 4) * 4;
    int b_tile_stride = BK / load_b_cache_time;

    float As_cache[2][TM] = {0.};  // double buffering
    float Bs_cache[2][TN] = {0.};  // double buffering

    int write_idx = 0;

#pragma unroll
    for (int i = 0; i < num_shared_block; ++i) {
#pragma unroll
        for (int m = 0; m < BM; m += a_tile_stride) {
            int cache_idx = m / a_tile_stride * 4;
            FETCH_FLOAT4(load_a_cache[cache_idx]) =
                    FETCH_FLOAT4(A[OFFSET(a_tile_row + m, a_tile_col, K)]);
            // Use load_a_cache for load 4 float at a time
            // As is saved as transpose matrix
            As[write_idx][a_tile_col][a_tile_row + m] = load_a_cache[cache_idx];
            // 这里 stride = 128，有 shared memory bank 冲突
            As[write_idx][a_tile_col + 1][a_tile_row + m] = load_a_cache[cache_idx + 1];
            As[write_idx][a_tile_col + 2][a_tile_row + m] = load_a_cache[cache_idx + 2];
            As[write_idx][a_tile_col + 3][a_tile_row + m] = load_a_cache[cache_idx + 3];
        }
#pragma unroll
        for (int k = 0; k < BK; k += b_tile_stride) {
            FETCH_FLOAT4(Bs[write_idx][b_tile_row + k][b_tile_col]) =
                    FETCH_FLOAT4(B[OFFSET(b_tile_row + k, b_tile_col, N)]);
        }
        __syncthreads();
        A += BK;    // Start position of next tile block to be processed
        B += BK * N;    // Start position of next tile block to be processed

#pragma unroll
        for (int k = 0; k < BK; ++k) {
#pragma unroll
//            for (int m = 0; m < TM; m += 4) {
            for (int m = 0, mm = 0; m < BM && mm < TM; m += block_row_thread * 4, mm += 4) {
//                int A_row = threadIdx.y * TM + m;
                int A_row = m + threadIdx.y * 4;
                FETCH_FLOAT4(As_cache[write_idx][mm]) = FETCH_FLOAT4(As[write_idx][k][A_row]);
            }
#pragma unroll
//            for (int n = 0; n < TN; n += 4) {
            for (int n = 0, nn = 0; n < BN && nn < TN; n += block_col_thread * 4, nn += 4) {
//                int B_col = threadIdx.x * TN + n;
                int B_col = n + threadIdx.x * 4;
                FETCH_FLOAT4(Bs_cache[write_idx][nn]) = FETCH_FLOAT4(Bs[write_idx][k][B_col]);
            }
#pragma unroll
            for (int m = 0; m < TM; ++m) {
#pragma unroll
                for (int n = 0; n < TN; ++n) {
                    accum[m][n] += As_cache[write_idx][m] * Bs_cache[write_idx][n];
                }
            }
        }
        write_idx ^= 1;
    }

#pragma unroll
    for (int m = 0; m < TM; m += 4) {
        int C_row = (m / 4) * (block_row_thread * 4) + threadIdx.y * 4;
#pragma unroll
        for (int n = 0; n < TN; n += 4) {
            int C_col = (n / 4) * (block_col_thread * 4) + threadIdx.x * 4;
#pragma unroll
            for (int i = 0; i < 4; ++i) {
                FETCH_FLOAT4(load_a_cache) = FETCH_FLOAT4(C[OFFSET(C_row + i, C_col, N)]);
                load_a_cache[0] = alpha * accum[m + i][n] + beta * load_a_cache[0];
                load_a_cache[1] = alpha * accum[m + i][n + 1] + beta * load_a_cache[1];
                load_a_cache[2] = alpha * accum[m + i][n + 2] + beta * load_a_cache[2];
                load_a_cache[3] = alpha * accum[m + i][n + 3] + beta * load_a_cache[3];
                FETCH_FLOAT4(C[OFFSET(C_row + i, C_col, N)]) = FETCH_FLOAT4(load_a_cache);
            }
        }
    }
}

void test_tile_2d_split_kernel(hipblasHandle_t handle, int M, int N, int K,
                               float alpha, float *A, float *B, float beta, float *C) {
    const int size = 16;
    const int tile_size = 8;
    const int BM = size * tile_size;
    const int BN = size * tile_size;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;
    dim3 block(size, size);
    dim3 grid(CEIL_DIV(N, BN), CEIL_DIV(M, BM)); // note: change M and N here
    tile_2d_split_kernel<BM, BN, BK, TM, TN><<<grid, block>>>(M, N, K, alpha, A, B, beta, C);
}


template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void tile_1d_split_kernel(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    const int block_row_thread = BN / TN;
    const int block_col_thread = BM / TM;
    const int thread_num = block_row_thread * block_col_thread;
    int num_shared_block = CEIL_DIV(K, BK);

    __shared__ float As[2][BK][BM];    // transpose shared A for avoid bank conflict, for double buffering
    __shared__ float Bs[2][BK][BN];    // for double buffering

    float accum[TM][TN] = {0.};

    const int load_a_cache_time = (BK * BM) / thread_num / 4;  // Each thread load 4 float
    const int load_b_cache_time = (BK * BN) / thread_num / 4;  // Each thread load 4 float

    float load_a_cache[4 * load_a_cache_time];
//    float load_a_cache[4];
//    float load_b_cache[4 * load_b_cache_time];

    A = &A[OFFSET(blockIdx.y * BM, 0, K)]; // Set block start position
    B = &B[OFFSET(0, blockIdx.x * BN, N)];
    C = &C[OFFSET(blockIdx.y * BM, blockIdx.x * BN, N)];

    int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
    int a_tile_row = thread_id / (BK / 4);
    int a_tile_col = thread_id % (BK / 4) * 4;
    int a_tile_stride = BM / load_a_cache_time;
//    printf("A tile row, col, stride %d, %d, %d", a_tile_row, a_tile_col, a_tile_stride);

    int b_tile_row = thread_id / (BN / 4);
    int b_tile_col = thread_id % (BN / 4) * 4;
    int b_tile_stride = BK / load_b_cache_time;

    float As_cache[2][TM] = {0.};  // double buffering
    float Bs_cache[2][TN] = {0.};  // double buffering

    int write_idx = 0;

#pragma unroll
    for (int i = 0; i < num_shared_block; ++i) {
#pragma unroll
        for (int m = 0; m < BM; m += a_tile_stride) {
            int cache_idx = m / a_tile_stride * 4;
            FETCH_FLOAT4(load_a_cache[cache_idx]) =
                    FETCH_FLOAT4(A[OFFSET(a_tile_row + m, a_tile_col, K)]);
            // Use load_a_cache for load 4 float at a time
            // As is saved as transpose matrix
            As[write_idx][a_tile_col][a_tile_row + m] = load_a_cache[cache_idx];
            // 这里 stride = 128，有 shared memory bank 冲突
            As[write_idx][a_tile_col + 1][a_tile_row + m] = load_a_cache[cache_idx + 1];
            As[write_idx][a_tile_col + 2][a_tile_row + m] = load_a_cache[cache_idx + 2];
            As[write_idx][a_tile_col + 3][a_tile_row + m] = load_a_cache[cache_idx + 3];
        }
#pragma unroll
        for (int k = 0; k < BK; k += b_tile_stride) {
            FETCH_FLOAT4(Bs[write_idx][b_tile_row + k][b_tile_col]) =
                    FETCH_FLOAT4(B[OFFSET(b_tile_row + k, b_tile_col, N)]);
        }
        __syncthreads();
        A += BK;    // Start position of next tile block to be processed
        B += BK * N;    // Start position of next tile block to be processed

#pragma unroll
        for (int k = 0; k < BK; ++k) {
#pragma unroll
//            for (int m = 0; m < TM; m += 4) {
            for (int m = 0, mm = 0; m < BM && mm < TM; m += block_row_thread * 4, mm += 4) {
//                int A_row = threadIdx.y * TM + m;
                int A_row = m + threadIdx.y * 4;
                FETCH_FLOAT4(As_cache[write_idx][mm]) = FETCH_FLOAT4(As[write_idx][k][A_row]);
            }
#pragma unroll
            for (int n = 0; n < TN; n += 4) {
//            for (int n = 0, nn = 0; n < BN && nn < TN; n += block_col_thread * 4, nn += 4) {
                int B_col = threadIdx.x * TN + n;
//                int B_col = n + threadIdx.x * 4;
                FETCH_FLOAT4(Bs_cache[write_idx][n]) = FETCH_FLOAT4(Bs[write_idx][k][B_col]);
            }
#pragma unroll
            for (int m = 0; m < TM; ++m) {
#pragma unroll
                for (int n = 0; n < TN; ++n) {
                    accum[m][n] += As_cache[write_idx][m] * Bs_cache[write_idx][n];
                }
            }
        }
        write_idx ^= 1;
    }

#pragma unroll
    for (int m = 0; m < TM; m += 4) {
        int ROW = m / 4;
        int C_row = ROW * (block_row_thread * 4) + threadIdx.y * 4;
#pragma unroll
        for (int n = 0; n < TN; n += 4) {
            int C_col = threadIdx.x * TN + n;
#pragma unroll
            for (int i = 0; i < 4; ++i) {
                FETCH_FLOAT4(load_a_cache) = FETCH_FLOAT4(C[OFFSET(C_row + i, C_col, N)]);
                load_a_cache[0] = alpha * accum[m + i][n] + beta * load_a_cache[0];
                load_a_cache[1] = alpha * accum[m + i][n + 1] + beta * load_a_cache[1];
                load_a_cache[2] = alpha * accum[m + i][n + 2] + beta * load_a_cache[2];
                load_a_cache[3] = alpha * accum[m + i][n + 3] + beta * load_a_cache[3];
                FETCH_FLOAT4(C[OFFSET(C_row + i, C_col, N)]) = FETCH_FLOAT4(load_a_cache);
            }
        }
    }
}

void test_tile_1d_split_kernel(hipblasHandle_t handle, int M, int N, int K,
                               float alpha, float *A, float *B, float beta, float *C) {
    const int size = 16;
    const int tile_size = 8;
    const int BM = size * tile_size;
    const int BN = size * tile_size;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;
    dim3 block(size, size);
    dim3 grid(CEIL_DIV(N, BN), CEIL_DIV(M, BM)); // note: change M and N here
    tile_1d_split_kernel<BM, BN, BK, TM, TN><<<grid, block>>>(M, N, K, alpha, A, B, beta, C);
}
